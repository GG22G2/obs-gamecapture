//
// Created by h6706 on 2023/9/18.
//



#include <hip/hip_runtime.h>
#include <iostream>
#include <hip/hip_runtime_api.h>



using namespace std;



// bgra格式像素
bool sumTotalPixel(unsigned char *dpix1, int width, int height) {

    unsigned char *hpix1;
    hipHostMalloc(&hpix1, width * height * sizeof(int), hipHostMallocDefault);
    hipMemcpy(hpix1, dpix1, width * height * sizeof(int), hipMemcpyDeviceToHost);


    int sum = 0;
    for (int i = 0; i < width * height * 4; i++) {
        sum += hpix1[0] ;
    }
    std::cout << sum << endl;

    hipHostFree(hpix1);

    return false;
}


